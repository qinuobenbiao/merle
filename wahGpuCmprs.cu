#include "hip/hip_runtime.h"
#include "wahGpu.cuh"
#include "moderngpu/kernel_load_balance.cuh"
#include "moderngpu/kernel_compact.cuh"

mgpu::mem_t<int> wahCntExcScan(const int *wah, size_t sz, mgpu::context_t &ctx) {
  mgpu::mem_t<int> res(sz + 1, ctx);
  mgpu::transform_scan<int>([=] MGPU_DEVICE (int index) {
    int val = wah[index];
    return val & 0x80000000 ? val & 0x3fffffff : 1;
  }, sz, res.data(), mgpu::plus_t<int>(),
    res.data() + sz, ctx);
  return res;
}

void wahAndExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] &= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

void wahOrExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] |= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

void wahXorExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] ^= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

mgpu::mem_t<int> wahDecomp(const int *wah, const int *excScan, size_t sz,
                           mgpu::context_t &ctx, int decompSz)
{
  using namespace mgpu;
  if (decompSz == 0)
    dtoh(&decompSz, excScan + sz, 1);
  mem_t<int> res(decompSz, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE (int idx, int , int, tuple<int> w) {
    int val = get<0>(w);
    val = val & 0x80000000 ? val << 1 >> 31 : val;
    val &= 0x7fffffff;
    resData[idx] = val;
  }, decompSz, excScan, sz, make_tuple(wah), ctx);
  return res;
}

// `dec` is decompressed, `enc` is compressed
mgpu::mem_t<int> wahEncOpDec(const int *enc, const int *excScan, size_t encSz,
                             const int *dec, size_t decSz, wahOp op,
                             mgpu::context_t &ctx, int encDecompSz) {
  using namespace mgpu;
  if (encDecompSz == 0)
    dtoh(&encDecompSz, excScan + encSz, 1);
  size_t resSz = max(decSz, size_t(encDecompSz));
  mem_t<int> res(resSz, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE (int decAt, int, int, tuple<int> w) {
    int val = get<0>(w);
    int decVal = decAt < decSz ? dec[decAt] : 0;
    val = val & 0x80000000 ? val << 1 >> 31 : val;
    if (op == wahOp::OR) val |= decVal;
    // else if (op == wahOp::AND) val &= decVal;
    else val ^= decVal;
    resData[decAt] = val & 0x7fffffff;
  }, encDecompSz, excScan, encSz,
    make_tuple(enc), ctx);
  if (decSz > encDecompSz)
    dtod(encDecompSz + resData, encDecompSz + dec, decSz - encDecompSz);
  return res;
}

mgpu::mem_t<int> wahEncAndDec(const int *enc, const int *excScan, size_t encSz,
                              const int *dec, size_t decSz, mgpu::context_t &ctx) {
  using namespace mgpu;
  mem_t<int> workCntScan(encSz + 1, ctx);
  transform_scan<int, scan_type_inc>([=] MGPU_DEVICE (int thId) {
    if (thId == 0) return 0;
    uint32_t encWah = enc[thId - 1];
    if ((encWah >> 30) != 3)
      return 1;
    return int(encWah & 0x3fffffff);
  }, encSz + 1, workCntScan.data(), plus_t<int>(), discard_iterator_t<int>(), ctx);
  int nRes; dtoh(&nRes, workCntScan.data() + encSz, 1);

  mem_t<int> res(nRes, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE(int idx, int, int rank, tuple<int, int> c) {
    int encWah = get<1>(c), out; // down[seg]
    uint32_t ty = (uint32_t)encWah >> 30;
    if (ty == 2) { // enc is 0-fill
      out = encWah;
    } else {
      int decAt = get<0>(c) + rank;
      if (decAt >= decSz) { resData[idx] = 0x80000001; return; }
      int decWah = dec[decAt];
      if (ty == 3) // enc is 1-fill
        out = decWah;
      else // enc is tail
        out = decWah & encWah;
    }
    resData[idx] = out;
  }, nRes, workCntScan.data(), encSz + 1,
    make_tuple(excScan, enc), ctx);
  return res;
}

// `enc` is in place
void wahEncNo1AndDec(int *enc, const int *excScan, size_t encSz, const int *dec,
                     size_t decSz, mgpu::context_t &ctx) {
  mgpu::transform([=] MGPU_DEVICE (int encAt) {
    int encWah = enc[encAt], decAt = excScan[encAt], out;
    if (decAt >= decSz)
      out = 0x80000001;
    else if (encWah & 0x80000000)
      out = encWah;
    else
      out = encWah & dec[decAt];
    enc[encAt] = out;
  }, encSz, ctx);
}

mgpu::mem_t<int> wahCompact(const int* wah, size_t wahSz, mgpu::context_t& context) {
  auto compact = mgpu::transform_compact(wahSz, context);
  int resCnt = compact.upsweep([s=(int)wahSz - 1, wah] MGPU_DEVICE (int i) {
    int val = wah[i];
    if (val == 0x7fffffff) val = 0xc0000001;
    if (val == 0) val = 0x80000001;
    if (!(val & 0x80000000))
      return true;
    if (i == s)
      return (val & 0x40000000) != 0;

    val &= 0xc0000000;
    int nxtVal = wah[i + 1];
    if (nxtVal == 0x7fffffff) nxtVal = 0xc0000001;
    if (nxtVal == 0) nxtVal = 0x80000001;
    nxtVal &= 0xc0000000;
    return nxtVal != val;
  });

  mgpu::mem_t<int> cntIncScan(wahSz, context);
  mgpu::transform_scan<int, mgpu::scan_type_inc>([=] MGPU_DEVICE (int index) {
    int v = wah[index];
    return v & 0x80000000 ? v & 0x3fffffff : 1;
  }, wahSz, cntIncScan.data(), mgpu::plus_t<int>(),
    mgpu::discard_iterator_t<int>(), context);

  mgpu::mem_t<int> res(resCnt, context);
  mgpu::mem_t<int> cntScanCompacted(resCnt, context);
  int *scanDat = cntIncScan.data(), *scanCpaDat = cntScanCompacted.data(),
      *resDat = res.data();
  compact.downsweep([=] MGPU_DEVICE (int destIdx, int srcIdx) {
    resDat[destIdx] = wah[srcIdx];
    scanCpaDat[destIdx] = scanDat[srcIdx];
  });

  mgpu::transform([=] MGPU_DEVICE (int i) {
    auto val = resDat[i];
    if (val == 0x7fffffff) val = 0xc0000001;
    if (val == 0) val = 0x80000001;
    if (!(val & 0x80000000))
      return;
    auto sum = scanCpaDat[i], prevSum = i == 0 ? 0 : scanCpaDat[i - 1];
    val &= 0xc0000000;
    val += (sum - prevSum);
    resDat[i] = val;
  }, resCnt, context);
  return res;
}

mgpu::mem_t<int> wahCompress(const int* dec, size_t wahSz, mgpu::context_t& context) {
  auto compact = mgpu::transform_compact(wahSz, context);
  int resCnt = compact.upsweep([s=(int)wahSz - 1, dec] MGPU_DEVICE (int i) {
    int val = dec[i];
    if (i == s) return val != 0;
    if (val != 0x7fffffff && val != 0) return true;
    return dec[i + 1] != val;
  });

  mgpu::mem_t<int> res(resCnt, context);
  mgpu::mem_t<int> cntScanCompacted(resCnt, context);
  int *scanCpaDat = cntScanCompacted.data(), *resDat = res.data();
  compact.downsweep([=] MGPU_DEVICE (int destIdx, int srcIdx) {
    resDat[destIdx] = dec[srcIdx];
    scanCpaDat[destIdx] = srcIdx + 1;
  });

  mgpu::transform([=] MGPU_DEVICE (int i) {
    auto val = resDat[i];
    if (val == 0x7fffffff) val = 0xc0000000;
    else if (val == 0) val = 0x80000000;
    else return;
    auto sum = scanCpaDat[i], prevSum = i == 0 ? 0 : scanCpaDat[i - 1];
    resDat[i] = val + sum - prevSum;
  }, resCnt, context);
  return res;
}
