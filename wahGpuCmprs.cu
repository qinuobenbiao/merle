#include "hip/hip_runtime.h"
#include "wahGpu.cuh"
#include "moderngpu/kernel_load_balance.cuh"
#include "moderngpu/kernel_compact.cuh"
#include <cstdint>

mgpu::mem_t<int> wahCntExcScan(const int *wah, size_t sz, mgpu::context_t &ctx) {
  mgpu::mem_t<int> res(sz + 1, ctx);
  mgpu::transform_scan<int>([=] MGPU_DEVICE (int index) {
    int val = wah[index];
    return val & 0x80000000 ? val & 0x3fffffff : 1;
  }, sz, res.data(), mgpu::plus_t<int>(),
    res.data() + sz, ctx);
  return res;
}

void wahAndExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] &= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

void wahOrExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] |= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

void wahXorExpl(int *upDecomp, const int *downDecomp, size_t minSz,
                mgpu::context_t &context) {
  mgpu::transform([] MGPU_DEVICE(int i, int *u, const int *d) { u[i] ^= d[i]; },
                  minSz, context, upDecomp, downDecomp);
}

mgpu::mem_t<int> wahDecomp(const int *wah, const int *excScan, size_t sz,
                           mgpu::context_t &ctx, int decompSz)
{
  using namespace mgpu;
  if (decompSz == 0)
    dtoh(&decompSz, excScan + sz, 1);
  mem_t<int> res(decompSz, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE (int idx, int , int, tuple<int> w) {
    int val = get<0>(w);
    val = val & 0x80000000 ? val << 1 >> 31 : val;
    val &= 0x7fffffff;
    resData[idx] = val;
  }, decompSz, excScan, sz, make_tuple(wah), ctx);
  return res;
}

// `dec` is decompressed, `enc` is compressed
mgpu::mem_t<int> wahEncOpDec(const int *enc, const int *excScan, size_t encSz,
                             const int *dec, size_t decSz, wahOp op,
                             mgpu::context_t &ctx, int encDecompSz) {
  using namespace mgpu;
  if (encDecompSz == 0)
    dtoh(&encDecompSz, excScan + encSz, 1);
  size_t resSz = max(decSz, size_t(encDecompSz));
  mem_t<int> res(resSz, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE (int decAt, int, int, tuple<int> w) {
    int val = get<0>(w);
    int decVal = decAt < decSz ? dec[decAt] : 0;
    val = val & 0x80000000 ? val << 1 >> 31 : val;
    if (op == wahOp::OR) val |= decVal;
    // else if (op == wahOp::AND) val &= decVal;
    else val ^= decVal;
    resData[decAt] = val & 0x7fffffff;
  }, encDecompSz, excScan, encSz,
    make_tuple(enc), ctx);
  if (decSz > encDecompSz)
    dtod(encDecompSz + resData, encDecompSz + dec, decSz - encDecompSz);
  return res;
}

mgpu::mem_t<int> wahEncAndDec(const int *enc, const int *excScan, size_t encSz,
                              const int *dec, size_t decSz, mgpu::context_t &ctx) {
  using namespace mgpu;
  mem_t<int> workCntScan(encSz + 1, ctx);
  transform_scan<int, scan_type_inc>([=] MGPU_DEVICE (int thId) {
    if (thId == 0) return 0;
    uint32_t encWah = enc[thId - 1];
    if ((encWah >> 30) != 3)
      return 1;
    return int(encWah & 0x3fffffff);
  }, encSz + 1, workCntScan.data(), plus_t<int>(), discard_iterator_t<int>(), ctx);
  int nRes; dtoh(&nRes, workCntScan.data() + encSz, 1);

  mem_t<int> res(nRes, ctx); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE(int idx, int, int rank, tuple<int, int> c) {
    int encWah = get<1>(c), out; // down[seg]
    uint32_t ty = (uint32_t)encWah >> 30;
    if (ty == 2) { // enc is 0-fill
      out = encWah;
    } else {
      int decAt = get<0>(c) + rank;
      if (decAt >= decSz) { resData[idx] = 0x80000001; return; }
      int decWah = dec[decAt];
      if (ty == 3) // enc is 1-fill
        out = decWah;
      else // enc is tail
        out = decWah & encWah;
    }
    resData[idx] = out;
  }, nRes, workCntScan.data(), encSz + 1,
    make_tuple(excScan, enc), ctx);
  return res;
}

// `enc` is in place
void wahEncNo1AndDec(int *enc, const int *excScan, size_t encSz, const int *dec,
                     size_t decSz, mgpu::context_t &ctx) {
  mgpu::transform([=] MGPU_DEVICE (int encAt) {
    int encWah = enc[encAt], decAt = excScan[encAt], out;
    if (decAt >= decSz)
      out = 0x80000001;
    else if (encWah & 0x80000000)
      out = encWah;
    else
      out = encWah & dec[decAt];
    enc[encAt] = out;
  }, encSz, ctx);
}

mgpu::mem_t<int> wahCompact(const int* wah, size_t wahSz, mgpu::context_t& context) {
  auto compact = mgpu::transform_compact(wahSz, context);
  int resCnt = compact.upsweep([s=(int)wahSz - 1, wah] MGPU_DEVICE (int i) {
    int val = wah[i];
    if (val == 0x7fffffff) val = 0xc0000001;
    if (val == 0) val = 0x80000001;
    if (!(val & 0x80000000))
      return true;
    if (i == s)
      return (val & 0x40000000) != 0;

    val &= 0xc0000000;
    int nxtVal = wah[i + 1];
    if (nxtVal == 0x7fffffff) nxtVal = 0xc0000001;
    if (nxtVal == 0) nxtVal = 0x80000001;
    nxtVal &= 0xc0000000;
    return nxtVal != val;
  });

  mgpu::mem_t<int> cntIncScan(wahSz, context);
  mgpu::transform_scan<int, mgpu::scan_type_inc>([=] MGPU_DEVICE (int index) {
    int v = wah[index];
    return v & 0x80000000 ? v & 0x3fffffff : 1;
  }, wahSz, cntIncScan.data(), mgpu::plus_t<int>(),
    mgpu::discard_iterator_t<int>(), context);

  mgpu::mem_t<int> res(resCnt, context);
  mgpu::mem_t<int> cntScanCompacted(resCnt, context);
  int *scanDat = cntIncScan.data(), *scanCpaDat = cntScanCompacted.data(),
      *resDat = res.data();
  compact.downsweep([=] MGPU_DEVICE (int destIdx, int srcIdx) {
    resDat[destIdx] = wah[srcIdx];
    scanCpaDat[destIdx] = scanDat[srcIdx];
  });

  mgpu::transform([=] MGPU_DEVICE (int i) {
    auto val = resDat[i];
    if (val == 0x7fffffff) val = 0xc0000001;
    if (val == 0) val = 0x80000001;
    if (!(val & 0x80000000))
      return;
    auto sum = scanCpaDat[i], prevSum = i == 0 ? 0 : scanCpaDat[i - 1];
    val &= 0xc0000000;
    val += (sum - prevSum);
    resDat[i] = val;
  }, resCnt, context);
  return res;
}

mgpu::mem_t<int> wahCompress(const int* dec, size_t wahSz, mgpu::context_t& context) {
  auto compact = mgpu::transform_compact(wahSz, context);
  int resCnt = compact.upsweep([s=(int)wahSz - 1, dec] MGPU_DEVICE (int i) {
    int val = dec[i];
    if (i == s) return val != 0;
    if (val != 0x7fffffff && val != 0) return true;
    return dec[i + 1] != val;
  });

  mgpu::mem_t<int> res(resCnt, context);
  mgpu::mem_t<int> cntScanCompacted(resCnt, context);
  int *scanCpaDat = cntScanCompacted.data(), *resDat = res.data();
  compact.downsweep([=] MGPU_DEVICE (int destIdx, int srcIdx) {
    resDat[destIdx] = dec[srcIdx];
    scanCpaDat[destIdx] = srcIdx + 1;
  });

  mgpu::transform([=] MGPU_DEVICE (int i) {
    auto val = resDat[i];
    if (val == 0x7fffffff) val = 0xc0000000;
    else if (val == 0) val = 0x80000000;
    else return;
    auto sum = scanCpaDat[i], prevSum = i == 0 ? 0 : scanCpaDat[i - 1];
    resDat[i] = val + sum - prevSum;
  }, resCnt, context);
  return res;
}

mgpu::mem_t<uint32_t>
dbjoinFlatWah(const uint32_t *fact, size_t factSz, const uint32_t* dim1,
              const uint32_t *dim2, uint32_t min, uint32_t max, mgpu::context_t &ctx) {
  // Each thread processes 32 elements, but 1 thread in each warp is idle
  // Each warp therefore processes 31 * 32 elements and produces 32 31bit words
  // Each cta processes 31 * 128 elements and produces 128 31bit words
  using launch_t = mgpu::launch_box_t<mgpu::arch_20_cta<128, 31>>;
  using params_t = launch_t::sm_ptx;
  static constexpr size_t nt = params_t::nt, vt = params_t::vt,
                          nv = nt * vt, vw = 32 * vt;
  const size_t resSz = factSz / 31;
  mgpu::mem_t<uint32_t> res(resSz, ctx);
  uint32_t* resData = (uint32_t*)res.data();

  auto sel_f = [=]MGPU_DEVICE(int tid, int cta) {
    size_t inWarpTid = tid % 32, elemIdx = nv * cta + vw * (tid / 32) + 30 - inWarpTid;
    uint32_t thrd_res = 999, cur_res; int bal;
    if (dim2 != nullptr) {
      #pragma unroll
      for (size_t i = 0; i < vt + 1; ++i, elemIdx += vt) {
        if (inWarpTid == 31 || elemIdx >= factSz) {
          cur_res = 0;
        } else {
          uint32_t val = dim2[dim1[fact[elemIdx]]];
          cur_res = val < max && val >= min;
        }
        cur_res = __ballot_sync(0x7fffffff, cur_res);
        if (inWarpTid == i)
          thrd_res = cur_res;
      }
    } else if (dim1 != nullptr) {
      #pragma unroll
      for (size_t i = 0; i < vt + 1; ++i, elemIdx += vt) {
        if (inWarpTid == 31 || elemIdx >= factSz) {
          bal = 0;
        } else {
          uint32_t val = dim1[fact[elemIdx]];
          bal = val < max && val >= min;
        }
        cur_res = __ballot_sync(0x7fffffff, bal);
        if (inWarpTid == i) {
          // if (cta == 0)
          //   printf("%u %d %zu %zu\n", cur_res, tid, inWarpTid, i);
          thrd_res = cur_res;
        }
      }
    } else {
      #pragma unroll
      for (size_t i = 0; i < vt + 1; ++i, elemIdx += vt) {
        if (inWarpTid == 31 || elemIdx >= factSz) {
          cur_res = 0;
        } else {
          uint32_t val = fact[elemIdx];
          cur_res = val < max && val >= min;
        }
        cur_res = __ballot_sync(0x7fffffff, cur_res);
        if (inWarpTid == i)
          thrd_res = cur_res;
      }
    }

    if (cta * nt + tid < resSz)
      resData[cta * nt + tid] = thrd_res;
  };
  mgpu::cta_transform<launch_t>(sel_f, factSz, ctx);
  return res;
}
