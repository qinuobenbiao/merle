#include "hip/hip_runtime.h"
#include "moderngpu/context.cuh"
#include "wahGpu.cuh"
#include <array>
using merle = mgpu::mem_t<int>;
// merle, merleXfer, decode-and-query, dnqXfer
using casetime = std::array<double, 5>;

extern "C" {
#include <malloc.h>
// nvcc hates intrinsics. Don't include <roaring.h> so that nvcc won't see the
// SSE/AVX intrinsics Roaring uses.
struct roaring_bitmap_t { char opaque[40]; };
size_t bitset_extract_setbits(uint64_t *bitset, size_t length, void *vout,
                              uint32_t base);
roaring_bitmap_t *roaring_bitmap_of_ptr(size_t n_args, const uint32_t *vals);
roaring_bitmap_t *roaring_bitmap_and(const roaring_bitmap_t *x1,
                                     const roaring_bitmap_t *x2);
roaring_bitmap_t *roaring_bitmap_or(const roaring_bitmap_t *x1,
                                    const roaring_bitmap_t *x2);
void roaring_bitmap_and_inplace(roaring_bitmap_t *x1,
                                const roaring_bitmap_t *x2);
uint64_t roaring_bitmap_get_cardinality(const roaring_bitmap_t *ra);
bool roaring_bitmap_run_optimize(roaring_bitmap_t *r);
void roaring_bitmap_free(const roaring_bitmap_t *r);

// roaring_bitmap_t *roaring_bitmap_of_devbitset(const mgpu::mem_t<int>& mem) {
roaring_bitmap_t *rbod(const mgpu::mem_t<int>& mem) {
  std::vector<int> bitset_h = from_mem(mem);
  std::vector<uint32_t> arr(bitset_h.size() * 31);
  size_t nr = bitset_extract_setbits((uint64_t *)bitset_h.data(),
                                     bitset_h.size() / 2, arr.data(), 0);
  arr.resize(nr);
  roaring_bitmap_t* res = roaring_bitmap_of_ptr(nr, arr.data());
  (void)roaring_bitmap_run_optimize(res);
  return res;
}
} // extern "C"

static std::pair<merle, merle> // wah and excscan
loadCase(int caseNr, int listNr, bool dec, mgpu::context_t &ctx) {
  char path[256];
  sprintf(path, "wahData/st/%dl%d.wah", caseNr, listNr);
  wahHost_s *h = loadWahFile(path);
  merle d = merle(h->cmprsNrWord, ctx);
  mgpu::htod(d.data(), (int*)h->dat, d.size());
  merle sc = wahCntExcScan(d.data(), d.size(), ctx);
  if (dec)
    d = wahDecomp(d.data(), sc.data(), d.size(), ctx);
  free(h);
  return std::make_pair(std::move(d), std::move(sc));
}

casetime t6_17(mgpu::context_t &ctx, bool is17) {
  int caseNr = is17 ? 17 : 6;
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(caseNr, 0, false, ctx);
  auto [l1d, l1sc] = loadCase(caseNr, 1, false, ctx);
  auto [l2d, l2sc] = loadCase(caseNr, 2, true, ctx);

  casetime ret{0, 0, 0, 0};
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = wahAndNo1(l1d.data(), l1sc.data(), l1d.size(), l0d.data(),
                          l0sc.data(), l0d.size(), ctx);
    wahEncNo1AndDec(res.data(), l0sc.data(), res.size(), l1d.data(),
                    l1d.size(), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    res = wahCompact(res.data(), res.size(), ctx);
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }

  ctx.timer_begin();
  l0d = wahDecomp(l0d.data(), l0sc.data(), l0d.size(), ctx);
  l1d = wahDecomp(l1d.data(), l1sc.data(), l1d.size(), ctx);
  ret[2] = ctx.timer_end() * 1000;
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = l2d.clone();
    wahAndExpl(res.data(), l0d.data(), res.size(), ctx);
    wahAndExpl(res.data(), l1d.data(), res.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d), *l2r = rbod(l2d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *res = roaring_bitmap_and(l0r, l1r);
    roaring_bitmap_and_inplace(res, l2r);
    roaring_bitmap_free(res);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r); roaring_bitmap_free(l2r);
  return ret;
}

casetime t3(mgpu::context_t &ctx) {
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(3, 0, true, ctx);
  auto [l1d, l1sc] = loadCase(3, 1, false, ctx);
  auto [l2d, l2sc] = loadCase(3, 2, true, ctx);
  auto [l3d, l3sc] = loadCase(3, 3, false, ctx);
  auto [l4d, l4sc] = loadCase(3, 4, true, ctx);

  casetime ret{0, 0, 0, 0};
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle or01 = wahEncOpDec(l1d.data(), l1sc.data(), l1d.size(), l0d.data(),
                             l0d.size(), wahOp::OR, ctx);
    merle or23 = wahEncOpDec(l3d.data(), l3sc.data(), l3d.size(), l2d.data(),
                             l2d.size(), wahOp::OR, ctx);
    wahAndExpl(or01.data(), or23.data(), std::min(or01.size(), or23.size()), ctx);
    wahAndExpl(or01.data(), l4d.data(), std::min(or01.size(), l4d.size()), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), or01.data(), or01.size());
    ret[1] += ctx.timer_end();
  }

  l1d = wahDecomp(l1d.data(), l1sc.data(), l1d.size(), ctx);
  l3d = wahDecomp(l3d.data(), l3sc.data(), l3d.size(), ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle or01 = l1d.clone(), or23 = l3d.clone();
    wahOrExpl(or01.data(), l0d.data(), or01.size(), ctx);
    wahOrExpl(or23.data(), l2d.data(), or23.size(), ctx);
    wahAndExpl(or01.data(), or23.data(), or01.size(), ctx);
    wahAndExpl(or01.data(), l4d.data(), or01.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), or01.data(), or01.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d), *l2r = rbod(l2d),
                   *l3r = rbod(l3d), *l4r = rbod(l4d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *or01 = roaring_bitmap_or(l0r, l1r);
    roaring_bitmap_t *or23 = roaring_bitmap_or(l2r, l3r);
    roaring_bitmap_and_inplace(or01, l4r);
    roaring_bitmap_and_inplace(or01, or23);
    roaring_bitmap_free(or01); roaring_bitmap_free(or23);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r); roaring_bitmap_free(l2r);
  roaring_bitmap_free(l3r); roaring_bitmap_free(l4r);
  return ret;
}

casetime t12(mgpu::context_t& ctx) {
  casetime ret{0, 0, 0, 0};
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(12, 0, true, ctx);
  auto [l1d, l1sc] = loadCase(12, 1, true, ctx);
  auto [l2d, l2sc] = loadCase(12, 2, false, ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle or01 = l0d.clone(), res = l2d.clone();
    wahOrExpl(or01.data(), l1d.data(), std::min(l0d.size(), l1d.size()), ctx);
    wahEncNo1AndDec(res.data(), l2sc.data(), res.size(), or01.data(), or01.size(), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }

  l2d = wahDecomp(l2d.data(), l2sc.data(), l2d.size(), ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = l0d.clone();
    wahOrExpl(res.data(), l1d.data(), res.size(), ctx);
    wahAndExpl(res.data(), l2d.data(), res.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d), *l2r = rbod(l2d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *res = roaring_bitmap_or(l0r, l1r);
    roaring_bitmap_and_inplace(res, l2r);
    roaring_bitmap_free(res);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r); roaring_bitmap_free(l2r);
  return ret;
}

casetime s12_13(mgpu::context_t& ctx, bool is3) {
  int caseNr = is3 ? -13 : -12;
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(caseNr, 0, true, ctx);
  auto [l1d, l1sc] = loadCase(caseNr, 1, true, ctx);
  auto [l2d, l2sc] = loadCase(caseNr, 2, false, ctx);

  casetime ret{0, 0, 0, 0};
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = wahEncAndDec(l2d.data(), l2sc.data(), l2d.size(), l0d.data(),
                             l0d.size(), ctx);
    wahEncNo1AndDec(res.data(), l2sc.data(), res.size(), l1d.data(),
                    l1d.size(), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }

  l2d = wahDecomp(l2d.data(), l2sc.data(), l2d.size(), ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = l2d.clone();
    wahAndExpl(res.data(), l0d.data(), res.size(), ctx);
    wahAndExpl(res.data(), l1d.data(), res.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d), *l2r = rbod(l2d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *res = roaring_bitmap_and(l0r, l2r);
    roaring_bitmap_and_inplace(res, l1r);
    roaring_bitmap_free(res);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r); roaring_bitmap_free(l2r);
  return ret;
}

casetime s23(mgpu::context_t& ctx) {
  casetime ret{0, 0, 0, 0};
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(-23, 0, false, ctx);
  auto [l1d, l1sc] = loadCase(-23, 1, true, ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = wahEncAndDec(l0d.data(), l0sc.data(), l0d.size(), l1d.data(),
                             l1d.size(), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }

  l0d = wahDecomp(l0d.data(), l0sc.data(), l0d.size(), ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res = l1d.clone();
    wahAndExpl(res.data(), l0d.data(), res.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *res = roaring_bitmap_and(l0r, l1r);
    roaring_bitmap_free(res);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r);
  return ret;
}

casetime s34(mgpu::context_t& ctx) {
  casetime ret{0, 0, 0, 0};
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(-34, 0, false, ctx);
  auto [l1d, l1sc] = loadCase(-34, 1, false, ctx);
  auto [l2d, l2sc] = loadCase(-34, 2, false, ctx);
  auto [l3d, l3sc] = loadCase(-34, 3, false, ctx);
  auto [l4d, l4sc] = loadCase(-34, 4, false, ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle or01 = wahOr(l1d.data(), l1sc.data(), l1d.size(), l0d.data(),
                       l0sc.data(), l0d.size(), ctx),
          or01s = wahCntExcScan(or01.data(), or01.size(), ctx);
    merle or23 = wahOr(l2d.data(), l2sc.data(), l2d.size(), l3d.data(),
                       l3sc.data(), l3d.size(), ctx),
          or23s = wahCntExcScan(or23.data(), or23.size(), ctx);
    merle res = wahAndNo1(or01.data(), or01s.data(), or01.size(), l4d.data(),
                          l4sc.data(), l4d.size(), ctx);
    assert(res.size() == l4d.size());
    res = wahAndNo1(or23.data(), or23s.data(), or23.size(), res.data(),
                    l4sc.data(), res.size(), ctx);
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }

  l0d = wahDecomp(l0d.data(), l0sc.data(), l0d.size(), ctx);
  l1d = wahDecomp(l1d.data(), l1sc.data(), l1d.size(), ctx);
  l2d = wahDecomp(l2d.data(), l2sc.data(), l2d.size(), ctx);
  l3d = wahDecomp(l3d.data(), l3sc.data(), l3d.size(), ctx);
  l4d = wahDecomp(l4d.data(), l4sc.data(), l4d.size(), ctx);
  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle or01 = l1d.clone(), or23 = l3d.clone();
    wahOrExpl(or01.data(), l0d.data(), or01.size(), ctx);
    wahOrExpl(or23.data(), l2d.data(), or23.size(), ctx);
    wahAndExpl(or01.data(), or23.data(), or01.size(), ctx);
    wahAndExpl(or01.data(), l4d.data(), or01.size(), ctx);
    ret[2] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), or01.data(), or01.size());
    ret[3] += ctx.timer_end();
  }

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d), *l2r = rbod(l2d),
                   *l3r = rbod(l3d), *l4r = rbod(l4d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *or01 = roaring_bitmap_or(l0r, l1r);
    roaring_bitmap_t *or23 = roaring_bitmap_or(l2r, l3r);
    roaring_bitmap_and_inplace(or01, l4r);
    roaring_bitmap_and_inplace(or01, or23);
    roaring_bitmap_free(or01); roaring_bitmap_free(or23);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r); roaring_bitmap_free(l2r);
  roaring_bitmap_free(l3r); roaring_bitmap_free(l4r);
  return ret;
}

casetime s41(mgpu::context_t& ctx) {
  casetime ret{0, 0, 0, 0};
  mgpu::mem_t<int> xferbuf(10 << 20, ctx, mgpu::memory_space_host);
  auto [l0d, l0sc] = loadCase(-41, 0, false, ctx);
  auto [l1d, l1sc] = loadCase(-41, 1, false, ctx);
  auto [l2d, l2sc] = loadCase(-41, 2, false, ctx);
  auto [l3d, l3sc] = loadCase(-41, 3, false, ctx);
  ctx.timer_begin();
  l0d = wahDecomp(l0d.data(), l0sc.data(), l0d.size(), ctx);
  l1d = wahDecomp(l1d.data(), l1sc.data(), l1d.size(), ctx);
  l2d = wahDecomp(l2d.data(), l2sc.data(), l2d.size(), ctx);
  l3d = wahDecomp(l3d.data(), l3sc.data(), l3d.size(), ctx);
  ret[2] = ctx.timer_end() * 1000;

  for (size_t i = 0; i < 1000; ++i) {
    ctx.timer_begin();
    merle res(l0d.size(), ctx);
    mgpu::transform(
        [] MGPU_DEVICE(int i, const int *l0, const int *l1, const int *l2,
                       const int *l3, int *r) {
          int o = l2[i], a = l0[i];
          o |= l3[i];
          a &= l1[i];
          a &= o;
          r[i] = a;
        },
        l0d.size(), ctx, l0d.data(), l1d.data(), l2d.data(), l3d.data(),
        res.data());
    ret[0] += ctx.timer_end();
    ctx.timer_begin();
    mgpu::dtoh(xferbuf.data(), res.data(), res.size());
    ret[1] += ctx.timer_end();
  }
  ret[2] += ret[0];
  ret[3] = ret[1];

  roaring_bitmap_t *l0r = rbod(l0d), *l1r = rbod(l1d),
                   *l2r = rbod(l2d), *l3r = rbod(l3d);
  ctx.timer_begin();
  for (size_t i = 0; i < 1000; i++) {
    roaring_bitmap_t *res = roaring_bitmap_or(l2r, l3r);
    roaring_bitmap_and_inplace(res, l0r);
    roaring_bitmap_and_inplace(res, l1r);
    roaring_bitmap_free(res);
  }
  ret[4] = ctx.timer_end();
  roaring_bitmap_free(l0r); roaring_bitmap_free(l1r);
  roaring_bitmap_free(l2r); roaring_bitmap_free(l3r);
  return ret;
}

void runTestCase(mgpu::context_t &ctx) {
#define bah(tcase)                                                             \
  printf(#tcase ",%f,%f,%f,%f,%f\n", t[0], t[1], t[2], t[3], t[4]);       \
  malloc_trim(0);
  puts("case,merle,xfer,dnq,xfer,roaring");
  casetime t;

  t = s12_13(ctx, false); bah(S12);
  t = s12_13(ctx, true); bah(S13);
  t = s23(ctx); bah(S23);
  t = s34(ctx); bah(S34);
  t = s41(ctx); bah(S41);
  t = t3(ctx); bah(T 3);
  t = t6_17(ctx, false); bah(T 6);
  t = t12(ctx); bah(T12);
  t = t6_17(ctx, true); bah(T17);
}
