#include "hip/hip_runtime.h"
#include "moderngpu/loadstore.cuh"
#include "wahGpu.cuh"
#include "moderngpu/kernel_scan.cuh"
#include "moderngpu/kernel_sortedsearch.cuh"
#include "moderngpu/kernel_load_balance.cuh"
using tplInt2 = mgpu::tuple<int, int>;

mgpu::mem_t<int> wahAndNo1(const int *up, const int *upIncScan, size_t upSz,
                           const int *down, const int *downIncScan,
                           size_t downSz, mgpu::context_t &context) {
  using namespace mgpu;
  using launch_t = mgpu::launch_box_t<mgpu::arch_20_cta<256, 10>>;
  mem_t<int> res(downSz, context); int* resData = res.data();
  auto lb = make_store_iterator<int>([=] MGPU_DEVICE (int upAt, int downAt) {
    int downWah = down[downAt], out;
    if (downWah == 0xc0000001) downWah = 0x7fffffff;
    if (upAt >= upSz) {
      out = 0x80000001;
    } else if (downWah & 0x80000000) { // 0-fill
      out = downWah;
    } else { // Tail: downVal == downWah
      int upVal = up[upAt];
      upVal = upVal & 0x80000000 ? int32_t(upVal) << 1 >> 31 : upVal;
      out = downWah & upVal;
      if (out == 0)
        out = 0x80000001;
    }
    resData[downAt] = out;
  });

  sorted_search<bounds_lower, launch_t>(downIncScan, downSz, upIncScan, upSz,
                                        lb, less_t<int>(), context);
  return res;
}

mgpu::mem_t<int> wahAnd(const int *up, const int *upExScan, size_t upSz,
                        const int *down, const int *downExScan, size_t downSz,
                        mgpu::context_t &context)
{
  using namespace mgpu;
  mem_t<int> lb(downSz + 1, context); int* lbData = lb.data();
  sorted_search_flag<bounds_lower>(downExScan, downSz + 1, upExScan, upSz + 1,
                                   lbData, mgpu::less_t<int>(), context);

  mem_t<int> workCntScan(downSz + 1, context);
  transform_scan<int, scan_type_inc>([=] MGPU_DEVICE (int thId) {
    if (thId == 0) return 0;
    if (((uint32_t)down[thId - 1] >> 30) != 3)
      return 1;
    int lbId = lbData[thId], prevLbId = lbData[thId - 1];
    return 0x7fffffff & int(lbId - prevLbId + 1 - (unsigned(prevLbId) >> 31));
  }, downSz + 1, workCntScan.data(), plus_t<int>(),
    discard_iterator_t<int>(), context);
  int nRes; dtoh(&nRes, workCntScan.data() + downSz, 1);

  mem_t<int> res(nRes, context); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE(int idx, int downAt /*seg*/, int rank, tplInt2 c) {
    int downWah = get<1>(c), out; // down[seg]
    uint32_t ty = (uint32_t)downWah >> 30;
    if (ty == 2)
      out = downWah;
    else {
      int upAt = get<0>(c), found = uint32_t(upAt) >> 31;
      upAt &= 0x7fffffff;
      upAt = upAt + rank - 1 + found;
      if (upAt >= upSz) { resData[idx] = 0x80000001; return; }
      int upWah = up[upAt];
      if (ty == 3) {  // down is 1-fill
        if (!(upWah & 0x80000000)) { // up is tail
          out = upWah;
        } else {
          int downBeg = downExScan[downAt],
              downEnd = downBeg + (downWah & 0x3fffffff),
              upBeg = upExScan[upAt], upEnd = upBeg + (upWah & 0x3fffffff),
              resBeg = rank == 0 ? downBeg : upBeg,
              resEnd = mgpu::min(upEnd, downEnd), outCnt = resEnd - resBeg;
          out = outCnt + (upWah & 0xc0000000);
        }
      } else { // down is tail
        int upVal = upWah & 0x80000000 ? int32_t(upWah) << 1 >> 31 : upWah;
        out = upVal & downWah;
      }
    }
    resData[idx] = out;
  },
  nRes, workCntScan.data(), downSz + 1,
  mgpu::make_tuple(lbData, down), context);
  return res;
}


template <int nt, int vt>
struct wahShm_t {
  int values[nt * vt];
  typename mgpu::cta_scan_t<nt, int>::storage_t scan;
};

template <int nt, int vt, int vt0, int vtmax>
MGPU_DEVICE void
innerDwnswp(int tid, int cta, mgpu::range_t cta_rg, const int *spine_red,
            const int *lb_dat, int *cntscan_dat, wahShm_t<nt, vtmax> &shared) {
  using namespace mgpu;
  // Load a tile to register in thread order.
  array_t<int, vt> x = mem_to_reg_strided<nt, vt, vt0, true>(
      lb_dat + cta_rg.begin, tid, cta_rg.count());
  reg_to_shared_strided<nt, vt>(x, tid, shared.values);
  thread_iterate<vt>([&](int i, int j) {
    x[i] = 1 - (unsigned(shared.values[j]) >> 31);
  }, tid);

  // Scan the unmatched flag with carry-in from the partials.
  // (clangd dies here for some reason and mistakenly emits errors)
  x = cta_scan_t<nt, int>().scan<vt, vt0>(
      tid, x, shared.scan, spine_red[cta], cta > 0, cta_rg.count(),
      plus_t<int>(), 0, scan_type_exc).scan;

  // Add scan result in threads to lb_dat in shared memory
  thread_iterate<vt>(
      [&](int i, int j) { (shared.values[j] += x[i]) &= 0x7fffffff; }, tid);
  __syncthreads();

  x = shared_to_reg_strided<nt, vt>(shared.values, tid, false);
  reg_to_mem_strided<nt, vt, vt0, true>(x, tid, cta_rg.count(),
                                        cntscan_dat + cta_rg.begin);

  // strided_iterate<nt, vt, vt0, true>( [&](int i, int j) { 
  //   cntscan_dat[j + cta_rg.begin] = shared.values[j]; 
  // }, tid, cta_rg.count());

  // shared_to_mem<nt, vt>(shared.values, tid, cta_rg.count(),
  //                       cntscan_dat + cta_rg.begin);
}

static std::pair<mgpu::mem_t<int>, mgpu::mem_t<int>>
pleaseWork(const int* needles, int num_needles, const int* haystack,
           int num_haystack, mgpu::context_t &context)
{
  using namespace mgpu;
  typedef launch_box_t<arch_20_cta<256, 11>> launch_t;
  typedef typename launch_t::sm_ptx params_t;
  enum { nt = params_t::nt, vt = params_t::vt, nv = nt * vt };

  // Partition the needles and haystacks into tiles.
  mem_t<int> partitions = merge_path_partitions<bounds_lower>(needles, num_needles,
    haystack, num_haystack, launch_t::nv(context), less_t<int>(), context);
  const int *mp_data = partitions.data();

  // Scan of unmatched counts in each cta
  cta_dim_t cta = launch_t::cta_dim(context.ptx_version());
  int num_ctas = div_up(num_haystack + num_needles, cta.nv());
  mem_t<int> spine_match(num_ctas + 1, context);
  int *spine_match_dat = spine_match.data();

  // Return value
  std::pair<mgpu::mem_t<int>, mgpu::mem_t<int>> ret = std::make_pair(
    mem_t<int>(num_needles, context),
    mem_t<int>(num_needles, context)
  );
  int* lb_dat = ret.first.data(), *cntscan_dat = ret.second.data();

  // Upsweep: produce both the lb (sorted_search_index) and reduces unmatch count
  // the first parts of this func is basically sorted_search_flag<bounds_lower>
  auto upswp_f = [=]MGPU_DEVICE(int tid, int cta) {
    __shared__ union {
      int keys[nt * (vt + 1)];
      int indices[nv];
      typename cta_reduce_t<nt, int>::storage_t reduce;
    } shared;

    // The range this CTA works on. Corresponds to a0 a1 b0 b1 in mgpu1
    merge_range_t cta_rg = compute_merge_range(
        num_needles, num_haystack, cta, nv, mp_data[cta], mp_data[cta + 1]);
    // aCount2 bCount2 in mgpu1
    int cta_nr_needle = cta_rg.a_count(), cta_nr_haystack = cta_rg.b_count(), out;
    if (cta_nr_needle != 0)
  { // don't want to indent this huge single branch

    // Equivalent to bounds_lower in sorted_search_flg
    bool // left_a = false,
        // left_b = bounds == bounds_upper && cta_rg.b_begin > 0,
        right_a = cta_rg.a_end < num_needles,
        right_b = cta_rg.b_end < num_haystack,
        extended = right_a && right_b; // && (bounds == bounds_lower || left_b);

    // Range of effective elements in shared memory including halos;
    merge_range_t shmem_rg = {
      .a_begin = 0 /*left_a*/, .a_end = cta_nr_needle + right_a,
      .b_begin = cta_nr_needle + right_a /*+ left_b*/,
      .b_end = cta_nr_needle + right_a /*+ left_b*/ + cta_nr_haystack + right_b
    };

    array_t<int, vt + 1> x = load_two_streams_reg_halo<nt, vt, int>(
      needles + cta_rg.a_begin /*-left_a*/, shmem_rg.a_end /*-a_begin*/,
      haystack + cta_rg.b_begin /*- left_b*/, shmem_rg.b_end - shmem_rg.a_end, tid
    );
    reg_to_shared_strided<nt, vt + 1>(x, tid, shared.keys);

    int diag = vt * tid;
    int mp = merge_path<bounds_lower>(shared.keys, cta_nr_needle,
        shared.keys + shmem_rg.b_begin, cta_nr_haystack, diag, less_t<int>());
    merge_range_t thread_rg = shmem_rg.partition(mp, diag);

    array_t<int, vt + 1> thrd_idx =
      extended ? serial_search_flg<vt, bounds_lower, false, true>(
          shared.keys, thread_rg, cta_rg.b_begin - shmem_rg.b_begin, less_t<int>()
      ) : serial_search_flg<vt, bounds_lower, true, true>(
          shared.keys, thread_rg, cta_rg.b_begin - shmem_rg.b_begin, less_t<int>()
      );
    // if (cta == 0)
    //   printf("tid %d acc %d\n", tid, thrd_idx[vt]);

    iterate<vt>([&] (int i) {
      if (thrd_idx[i] != -1)
        shared.indices[mp++] = thrd_idx[i];
    }); // lower bound register to shared
    __syncthreads();
    shared_to_mem<nt, vt>(shared.indices, tid, cta_nr_needle,
                          lb_dat + cta_rg.a_begin, true);

    int all_reduce = cta_reduce_t<nt, int>().reduce(tid, thrd_idx[vt],
        shared.reduce, nt, plus_t<int>(), false);
    out = cta_nr_needle - all_reduce;
  } else out = 0;
    if (tid == 0)
      spine_match_dat[cta] = out;
  };

  cta_launch<launch_t>(upswp_f, num_ctas, context);
  // Spine phase: Scan unmatch count
  scan<scan_type_exc>(spine_match_dat, num_ctas, spine_match_dat, plus_t<int>(),
                      spine_match_dat + num_ctas, context);

  // Downsweep phase
  auto downswp_f = [=]MGPU_DEVICE(int tid, int cta) {
    __shared__ wahShm_t<nt, vt> shared;
    range_t cta_rg = compute_merge_range(num_needles, num_haystack,
        cta, nv, mp_data[cta], mp_data[cta + 1]).a_range();
    if (cta_rg.count() <= 0)
      return;

    if (cta_rg.count() < nt * 4) {
      innerDwnswp<nt, 4, 0, vt>(tid, cta, cta_rg, spine_match_dat, lb_dat,
                                 cntscan_dat, shared);
    } else if (cta_rg.count() < nt * 6) {
      innerDwnswp<nt, 6, 4, vt>(tid, cta, cta_rg, spine_match_dat, lb_dat,
                                 cntscan_dat, shared);
    } else if (cta_rg.count() < nt * 8) {
      innerDwnswp<nt, 8, 6, vt>(tid, cta, cta_rg, spine_match_dat, lb_dat,
                                 cntscan_dat, shared);
    } else {
      innerDwnswp<nt, vt, 8, vt>(tid, cta, cta_rg, spine_match_dat, lb_dat,
                                  cntscan_dat, shared);
    }
  };
  cta_launch<launch_t>(downswp_f, num_ctas, context);
  int nRes; dtoh(&nRes, cntscan_dat + num_needles - 1, 1);
  // std::cout << "pleaseWork reduction " << nRes << std::endl;
  return ret;
}


mgpu::mem_t<int> wahOr(const int *up, const int *upExScan, size_t upSz,
                       const int *down, const int *downExScan, size_t downSz,
                       mgpu::context_t &context)
{
  using namespace mgpu;
  auto [lb, workCntScan] =
      pleaseWork(downExScan, downSz + 1, upExScan, upSz + 1, context);
  int* lbData = lb.data();
  int nRes; dtoh(&nRes, workCntScan.data() + downSz, 1);
  // std::cout << nRes << std::endl;

  mem_t<int> res(nRes, context); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE(int idx, int downAt, int rank, tplInt2 c) {
    int upAt = get<0>(c), found = uint32_t(upAt) >> 31;
    int downWah = get<1>(c), out; // down[seg]
    upAt = (upAt & 0x7fffffff) + rank - 1 + found;
    if (upAt >= upSz) {
      if (rank != 0) // will only run at most once
        resData[idx] =
            downExScan[downAt + 1] - upExScan[upSz] + (downWah & 0xc0000000);
      else resData[idx] = downWah;
      return;
    }
    int upWah = up[upAt];
    if (((uint32_t)downWah >> 31) == 0) { // down is tail
      int upVal = upWah & 0x80000000 ? int32_t(upWah) << 1 >> 31 : upWah;
      out = (upVal | downWah) & 0x7fffffff;
    } else if (((uint32_t)upWah >> 31) == 0) { // up is tail, down is fill
      int downVal = downWah << 1 >> 31;
      out = (downVal | upWah) & 0x7fffffff;
    } else { // both are fills
      int downBeg = downExScan[downAt],
          downEnd = downBeg + (downWah & 0x3fffffff), upBeg = upExScan[upAt],
          upEnd = upBeg + (upWah & 0x3fffffff),
          resBeg = rank == 0 ? downBeg : upBeg,
          resEnd = mgpu::min(upEnd, downEnd), outCnt = resEnd - resBeg;
      out = outCnt + ((upWah | downWah) & 0xc0000000);
    }
    resData[idx] = out;
  },
  nRes, workCntScan.data(), downSz + 1,
  mgpu::make_tuple(lbData, down), context);
  return res;
}

mgpu::mem_t<int> wahXor(const int *up, const int *upExScan, size_t upSz,
                        const int *down, const int *downExScan, size_t downSz,
                        mgpu::context_t &context)
{
  using namespace mgpu;
  auto [lb, workCntScan] =
      pleaseWork(downExScan, downSz + 1, upExScan, upSz + 1, context);
  int* lbData = lb.data();
  int nRes; dtoh(&nRes, workCntScan.data() + downSz, 1);

  mem_t<int> res(nRes, context); int* resData = res.data();
  transform_lbs([=] MGPU_DEVICE(int idx, int downAt, int rank, tplInt2 c) {
    int upAt = get<0>(c), found = uint32_t(upAt) >> 31;
    int downWah = get<1>(c), out; // down[seg]
    upAt = (upAt & 0x7fffffff) + rank - 1 + found;
    if (upAt >= upSz) {
      if (rank != 0) // will only run at most once
        resData[idx] =
            downExScan[downAt + 1] - upExScan[upSz] + (downWah & 0xc0000000);
      else resData[idx] = downWah;
      return;
    }
    int upWah = up[upAt];
    if (((uint32_t)downWah >> 31) == 0) { // down is tail
      int upVal = upWah & 0x80000000 ? int32_t(upWah) << 1 >> 31 : upWah;
      out = (upVal ^ downWah) & 0x7fffffff;
    } else if (((uint32_t)upWah >> 31) == 0) { // up is tail, down is fill
      int downVal = downWah << 1 >> 31;
      out = (downVal ^ upWah) & 0x7fffffff;
    } else { // both are fills
      int downBeg = downExScan[downAt],
          downEnd = downBeg + (downWah & 0x3fffffff), upBeg = upExScan[upAt],
          upEnd = upBeg + (upWah & 0x3fffffff),
          resBeg = rank == 0 ? downBeg : upBeg,
          resEnd = mgpu::min(upEnd, downEnd), outCnt = resEnd - resBeg;
      out = outCnt + ((upWah ^ downWah) & 0x40000000) + 0x80000000;
    }
    resData[idx] = out;
  },
  nRes, workCntScan.data(), downSz + 1,
  mgpu::make_tuple(lbData, down), context);
  return res;
}
