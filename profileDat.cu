#include "hip/hip_runtime.h"
#include "wahGpu.cuh"

wahStashDat_s::wahStashDat_s(size_t id_, wahDev_s &wah, size_t nIter)
    : id(id_), execScanTime(0), decTime(0) {
  for (size_t i = 0; i < nIter; ++i) {
    execScanTime += wah.timeExcScan();
    decTime += wah.timeDecompress();
  }
  execScanTime /= nIter;
  decTime /= nIter;
  cmprsNrWord = wah._wah.size();
  decmNrWord = wah._decomp.size();
}

void wahOpProfile::profileAnd(bool semi) {
  explOpTime = lhsDev.timeExplAnd(rhsDev);
  explXferTime = lhsDev.timeXfer();
  // explCrsTime = lhsDev.timeCompress();
  // explCrsXferTime = lhsDev.timeXfer();

  // lhsDev.context().timer_begin();
  // uint32_t* out = new uint32_t[lhsDev._opResH.size() + 60];
  // wahAndCPU(lhsHost->dat, lhsHost->dat + lhsHost->cmprsNrWord, rhsHost->dat,
  //           rhsHost->dat + rhsHost->cmprsNrWord, out,
  //           out + lhsDev._opResH.size() + 50);
  // cpuTime = lhsDev.context().timer_end();
  // if (memcmp(lhsDev._opResH.data(), out, lhsDev._opResH.size()) != 0)
  //   exit(fputs("WAH explicit AND error!\n", stderr));

  directOpTime = rhsDev.timeDirectAnd(lhsDev);
  directXferTime = rhsDev.timeXfer();
  // directCptTime = rhsDev.timeCompact();
  // directCptXferTime = rhsDev.timeXfer();

  // if (memcmp(rhsDev._opResH.data(), out, rhsDev._opResH.size()) != 0)
  //   exit(fputs("WAH direct AND error!\n", stderr));
  // delete[] out;
  lhsDev._opResD = mgpu::mem_t<int>();
  rhsDev._opResD = mgpu::mem_t<int>();

  if (!semi) return;
  auto hostRes = std::move(lhsDev._opResH);
  wahDev_s& larg = lhsDev._wah.size() > rhsDev._wah.size() ? lhsDev : rhsDev;
  wahDev_s& smal = lhsDev._wah.size() > rhsDev._wah.size() ? rhsDev : lhsDev;
  semiOpTime = larg.timeSemiExplAnd(smal);
  semiXferTime = larg.timeXfer();
  // semiCrsTime = larg.timeCompact();
  // semiCrsXferTime = larg.timeXfer();
  // if (hostRes != larg._opResH)
  //   exit(fputs("WAH semi AND error!\n", stderr));
  larg._opResD = mgpu::mem_t<int>();
}

void wahOpProfile::profileOr(bool semi) {
  explOpTime = lhsDev.timeExplOr(rhsDev);
  explXferTime = lhsDev.timeXfer();
  // explCrsTime = lhsDev.timeCompress();
  // explCrsXferTime = lhsDev.timeXfer();

  // TODO: Currently the CPU AND also serves as a baseline for GPU OR since in
  // branchless CPU implementation, all logics in AND, OR, XOR are virtually
  // identical except that the extra part of longer sequence is discarded.
  // lhsDev.context().timer_begin();
  // uint32_t* out = new uint32_t[lhsDev._opResH.size() + 60];
  // wahAndCPU(lhsHost->dat, lhsHost->dat + lhsHost->cmprsNrWord, rhsHost->dat,
  //           rhsHost->dat + rhsHost->cmprsNrWord, out,
  //           out + lhsDev._opResH.size() + 50);
  // cpuTime = lhsDev.context().timer_end() * 1.1;
  // delete[] out;

  directOpTime = rhsDev.timeDirectOr(lhsDev);
  directXferTime = rhsDev.timeXfer();
  // directCptTime = rhsDev.timeCompact();
  // directCptXferTime = rhsDev.timeXfer();
  // if (lhsDev._opResH != rhsDev._opResH)
  //   exit(fputs("WAH OR error!\n", stderr));
  lhsDev._opResD = mgpu::mem_t<int>();
  rhsDev._opResD = mgpu::mem_t<int>();

  if (!semi) return;
  auto hostRes = std::move(lhsDev._opResH);
  wahDev_s& larg = lhsDev._wah.size() > rhsDev._wah.size() ? lhsDev : rhsDev;
  wahDev_s& smal = lhsDev._wah.size() > rhsDev._wah.size() ? rhsDev : lhsDev;
  semiOpTime = larg.timeSemiExplOr(smal);
  semiXferTime = larg.timeXfer();
  // semiCrsTime = larg.timeCompress();
  // semiCrsXferTime = larg.timeXfer();
  // if (hostRes != larg._opResH)
  //   exit(fputs("WAH semi OR error!\n", stderr));
  larg._opResD = mgpu::mem_t<int>();
}

void wahOpProfile::profileXor(bool semi) {
  explOpTime = lhsDev.timeExplXor(rhsDev);
  explXferTime = lhsDev.timeXfer();
  // explCrsTime = lhsDev.timeCompress();
  // explCrsXferTime = lhsDev.timeXfer();

  // lhsDev.context().timer_begin();
  // uint32_t* out = new uint32_t[lhsDev._opResH.size() + 60];
  // wahAndCPU(lhsHost->dat, lhsHost->dat + lhsHost->cmprsNrWord, rhsHost->dat,
  //           rhsHost->dat + rhsHost->cmprsNrWord, out,
  //           out + lhsDev._opResH.size() + 50);
  // cpuTime = lhsDev.context().timer_end() * 1.1;
  // delete[] out;

  directOpTime = rhsDev.timeDirectXor(lhsDev);
  directXferTime = rhsDev.timeXfer();
  // directCptTime = rhsDev.timeCompact();
  // directCptXferTime = rhsDev.timeXfer();
  // if (lhsDev._opResH != rhsDev._opResH)
  //   exit(fputs("WAH XOR error!\n", stderr));
  lhsDev._opResD = mgpu::mem_t<int>();
  rhsDev._opResD = mgpu::mem_t<int>();

  if (!semi) return;
  auto hostRes = std::move(lhsDev._opResH);
  wahDev_s& larg = lhsDev._wah.size() > rhsDev._wah.size() ? lhsDev : rhsDev;
  wahDev_s& smal = lhsDev._wah.size() > rhsDev._wah.size() ? rhsDev : lhsDev;
  semiOpTime = larg.timeSemiExplXor(smal);
  semiXferTime = larg.timeXfer();
  // semiCrsTime = larg.timeCompress();
  // semiCrsXferTime = larg.timeXfer();
  // if (hostRes != larg._opResH)
  //   exit(fputs("WAH semi XOR error!\n", stderr));
  larg._opResD = mgpu::mem_t<int>();
}

void wahOpProfile::profile(wahOp op, bool semi) {
  if (op == wahOp::AND)
    profileAnd(semi);
  else if (op == wahOp::OR)
    profileOr(semi);
  else profileXor(semi);
}

double wahOpProfile::explTotalTime(int lStash, int rStash) const noexcept {
  double r = mgpu::min(explXferTime, explCrsTime + explCrsXferTime) + explOpTime;
  if (lStash < 3) r += lhsDat.decTime;
  if (rStash < 3) r += rhsDat.decTime;
  if (lStash < 2) r += lhsDat.execScanTime;
  if (rStash < 2) r += rhsDat.execScanTime;
  return r;
}

double wahOpProfile::directTotalTime(int lStash, int rStash) const noexcept {
  double r = mgpu::min(directXferTime, directCptTime + directCptXferTime) +
             directOpTime;
  if (lStash < 2) r += lhsDat.execScanTime;
  if (rStash < 2) r += rhsDat.execScanTime;
  return r;
}

double wahOpProfile::semiTotalTime() const noexcept {
  return mgpu::min(semiXferTime, semiCrsTime + semiCrsXferTime) + semiOpTime;
}

void wahOpProfile::setLhs(mgpu::context_t &ctx, const wahHost_s *lhsH,
                          wahStashDat_s dat) {
  lhsDat = dat;
  lhsHost = lhsH;
  lhsDev.~wahDev_s();
  new (&lhsDev) wahDev_s(ctx, lhsH);
}

void wahOpProfile::setRhs(mgpu::context_t &ctx, const wahHost_s *rhsH,
                          wahStashDat_s dat) {
  rhsDat = dat;
  rhsHost = rhsH;
  rhsDev.~wahDev_s();
  new (&rhsDev) wahDev_s(ctx, rhsH);
}

void wahOpProfile::print(FILE* stream, bool semi) const noexcept {
  if (semi)
    fprintf(stream,
        "%zu,%zu,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f\n",
        lhsDat.id, rhsDat.id, explOpTime, explXferTime,
        explCrsTime + explCrsXferTime, directOpTime, directXferTime,
        directCptTime + directCptXferTime, semiOpTime, semiXferTime,
        semiCrsTime + semiCrsXferTime, cpuTime);
  else
    fprintf(stream, "%zu,%zu,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f,%.06f\n",
            lhsDat.id, rhsDat.id, explOpTime, explXferTime,
            explCrsTime + explCrsXferTime, directOpTime, directXferTime,
            directCptTime + directCptXferTime, cpuTime);
}

void wahStashDat_s::print(FILE* stream) const noexcept {
  fprintf(stream, "%zu,%.06f,%.06f,%zu,%zu\n", id, stash2Time(), stash3Time(),
          stash2Sz(), stash3Sz());
}

void wahOpProfile::clear() {
  lhsDev = wahDev_s();
  rhsDev = wahDev_s();
  lhsHost = nullptr;
  rhsHost = nullptr;
}
